#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// CUDA kernel 
__global__ void VectorAdd(int *a, int *b, int *c, int length){
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    if ( tId< length){
        c[tId] = a[tId] + b[tId];
    }
}

int main(){
    // Get vector length
	int userInput;
    printf("Enter the length of the vectors in range of hundreds and thousands: ");
    scanf("%d", &userInput);

    int *h_a, *h_b, *h_c;   // initialize host vectors 
    int *d_a, *d_b, *d_c;   // initialize device vectors 
    int size = userInput * sizeof(int);

    h_a = (int *)malloc(size);
    h_b = (int *)malloc(size);
    h_c = (int *)malloc(size);

    // calculate grid and vector size 
    int blockSize = 256;
    int numBlocks = (userInput + blockSize - 1) / blockSize;  

    // initialize vectors on host
    for (int i = 0; i < userInput; ++i) {
        h_a[i] = h_b[i] = i;
    }

    // GPU memory allocation
    gpuErrchk(hipMalloc((void **)&d_a, size));
    gpuErrchk(hipMalloc((void **)&d_b, size));
    gpuErrchk(hipMalloc((void **)&d_c, size));

    // Copy to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch the kernel
    VectorAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, userInput);

    // Copy the array back to host 
    gpuErrchk(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));

    printf("Vector addition result:\n");
    for (int i = 0; i < userInput; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free GPU memory 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free CPU memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;


}
